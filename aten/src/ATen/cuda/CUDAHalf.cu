#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "ATen/core/Half.h"
#include "ATen/cuda/CUDAHalf.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

namespace at {
#if CUDA_VERSION < 9000 && !defined(__HIP_PLATFORM_HCC__)

half Converter<half, Half>::operator()(Half aten_half) {
  return half{aten_half.x};
}

half Converter<half, double>::operator()(double value) {
  return half{Half(value).x};
}

Half Converter<Half, half>::operator()(half cuda_half) {
  return Half(cuda_half.x, Half::from_bits);
}
#else
half Converter<half, Half>::operator()(Half aten_half) {
  __half_raw x_raw;
  x_raw.x = aten_half.x;
  return half(x_raw);
}

Half Converter<Half, half>::operator()(half cuda_half) {
  __half_raw raw(cuda_half);
  return Half(raw.x, Half::from_bits);
}

half Converter<half, double>::operator()(double value) {
  __half_raw raw;
  raw.x = Half(value).x;
  return half {raw};
}

template <> __half HalfFix(Half h) {
  __half_raw raw;
  raw.x = h.x;
  return __half{raw};
}

template <> Half HalfFix(__half h) {
  __half_raw raw(h);
  return Half(raw.x, Half::from_bits);
}
#endif
} // namespace at
