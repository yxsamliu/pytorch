#include "hip/hip_runtime.h"
#include "THCBlas.h"
#include "THCGeneral.h"
#include "TH/THHalf.h"

#include <algorithm>

float THCudaBlas_Sdot(THCState *state, int64_t n, float *x, int64_t incx, float *y, int64_t incy)
{
  if (n == 1) {
    incx = 1;
    incy = 1;
  }

  if ((n <= INT_MAX) && (incx <= INT_MAX) && (incy <= INT_MAX)) {
    int i_n = (int)n;
    int i_incx = (int)incx;
    int i_incy = (int)incy;
    float result;
    rocblas_handle handle = THCState_getCurrentBlasHandle(state);
    rocblas_set_stream(handle, THCState_getCurrentStream(state));
    THCublasCheck(rocblas_sdot(handle, i_n, x, i_incx, y, i_incy, &result));
    return result;
  }

  THError("Cublas_Sdot only supports n, incx and incy "
          "up to signed integer limits: %d", INT_MAX);
  return 0;
}

double THCudaBlas_Ddot(THCState *state, int64_t n, double *x, int64_t incx, double *y, int64_t incy)
{
  if (n == 1) {
    incx = 1;
    incy = 1;
  }

  if ((n <= INT_MAX) && (incx <= INT_MAX) && (incy <= INT_MAX)) {
    int i_n = (int)n;
    int i_incx = (int)incx;
    int i_incy = (int)incy;
    double result;
    rocblas_handle handle = THCState_getCurrentBlasHandle(state);
    rocblas_set_stream(handle, THCState_getCurrentStream(state));
    THCublasCheck(rocblas_ddot(handle, i_n, x, i_incx, y, i_incy, &result));
    return result;
  }

  THError("Cublas_Ddot only supports n, incx and incy "
          "up to signed integer limits: %d", INT_MAX);
  return 0;
}

at::Half THCudaBlas_Hdot(THCState *state, int64_t n, at::Half *x, int64_t incx, at::Half *y, int64_t incy)
{
#if CUDA_VERSION >= 8000
  if (n == 1) {
    incx = 1;
    incy = 1;
  }

  if ((n <= INT_MAX) && (incx <= INT_MAX) && (incy <= INT_MAX)) {
    at::Half result;
    rocblas_handle handle = THCState_getCurrentBlasHandle(state);
    rocblas_set_stream(handle, THCState_getCurrentStream(state));
    THCublasCheck(rocblas_dotex(handle, n,
                              x, hipR16F, incx,
                              y, hipR16F, incy,
                              &result, hipR16F,
                              hipR32F));
    return result;
  }

  THError("Cublas_Hdot only supports n, incx and incy "
          "up to signed integer limits: %d", INT_MAX);
  return 0.0;
#else
  THError("Cublas_Hdot requires CUDA 8.0+");
  return 0.0;
#endif
}

/* Level 2 */

void adjustLdLevel2(int64_t m, int64_t n, int64_t *lda)
{
  // Note: leading dimensions generally are checked that they are > 0 and at least as big the result
  // requires (even if the value won't be used).
  // TODO: why does Level3 check trans but this doesn't?
  if (n <= 1)
    *lda = std::max<int64_t>(m, 1);
}

void THCudaBlas_Sgemv(THCState *state, char trans, int64_t m, int64_t n, float alpha, float *a, int64_t lda, float *x, int64_t incx, float beta, float *y, int64_t incy)
{
  adjustLdLevel2(m, n, &lda);

  rocblas_operation op;
  if (trans == 't') op = rocblas_operation_transpose;
  else if (trans == 'n') op = rocblas_operation_none;
  else if (trans == 'c') op = rocblas_operation_conjugate_transpose;
  else THError("Cublas_Sgemv parameter trans should be 't', 'n' or 'c'.");

  if( (m <= INT_MAX) && (n <= INT_MAX) &&
      (lda > 0) && (lda <= INT_MAX) &&
      (incx > 0) && (incx <= INT_MAX) &&
      (incy > 0) && (incy <= INT_MAX) )
  {
    int i_m = (int)m;
    int i_n = (int)n;
    int i_lda = (int)lda;
    int i_incx = (int)incx;
    int i_incy = (int)incy;

    rocblas_handle handle = THCState_getCurrentBlasHandle(state);
    rocblas_set_stream(handle, THCState_getCurrentStream(state));
    THCublasCheck(rocblas_sgemv(handle, op, i_m, i_n, &alpha, a, i_lda, x, i_incx, &beta, y, i_incy));
    return;
  }
  THError("Cublas_Sgemv only supports m, n, lda, incx, incy"
          "in the range 0 < [val] <= %d", INT_MAX);
}

void THCudaBlas_Dgemv(THCState *state, char trans, int64_t m, int64_t n, double alpha, double *a, int64_t lda, double *x, int64_t incx, double beta, double *y, int64_t incy)
{
  adjustLdLevel2(m, n, &lda);

  rocblas_operation op;
  if (trans == 't') op = rocblas_operation_transpose;
  else if (trans == 'n') op = rocblas_operation_none;
  else if (trans == 'c') op = rocblas_operation_conjugate_transpose;
  else THError("Cublas_Sgemv parameter trans should be 't', 'n' or 'c'.");

  if( (m <= INT_MAX) && (n <= INT_MAX) &&
      (lda > 0) && (lda <= INT_MAX) &&
      (incx > 0) && (incx <= INT_MAX) &&
      (incy > 0) && (incy <= INT_MAX) )
  {
    int i_m = (int)m;
    int i_n = (int)n;
    int i_lda = (int)lda;
    int i_incx = (int)incx;
    int i_incy = (int)incy;

    rocblas_handle handle = THCState_getCurrentBlasHandle(state);
    rocblas_set_stream(handle, THCState_getCurrentStream(state));
    THCublasCheck(rocblas_dgemv(handle, op, i_m, i_n, &alpha, a, i_lda, x, i_incx, &beta, y, i_incy));
    return;
  }
  THError("Cublas_Dgemv only supports m, n, lda, incx, incy"
          "in the range 0 < [val] <= %d", INT_MAX);
}

void THCudaBlas_Sger(THCState *state, int64_t m, int64_t n, float alpha, float *x, int64_t incx, float *y, int64_t incy, float *a, int64_t lda)
{
  adjustLdLevel2(m, n, &lda);

  if( (m <= INT_MAX) && (n <= INT_MAX) && (lda <= INT_MAX)  && (incx <= INT_MAX) && (incy <= INT_MAX) )
    {
      int i_m = (int)m;
      int i_n = (int)n;
      int i_lda = (int)lda;
      int i_incx = (int)incx;
      int i_incy = (int)incy;

      rocblas_handle handle = THCState_getCurrentBlasHandle(state);
      rocblas_set_stream(handle, THCState_getCurrentStream(state));
      THCublasCheck(rocblas_sger(handle, i_m, i_n, &alpha, x, i_incx, y, i_incy, a, i_lda));
      return;
    }
  THError("Cublas_Sger only supports m, n, lda, incx, incy"
          "with the bound [val] <= %d", INT_MAX);
}

void THCudaBlas_Dger(THCState *state, int64_t m, int64_t n, double alpha, double *x, int64_t incx, double *y, int64_t incy, double *a, int64_t lda)
{
  adjustLdLevel2(m, n, &lda);

  if( (m <= INT_MAX) && (n <= INT_MAX) && (lda <= INT_MAX)  && (incx <= INT_MAX) && (incy <= INT_MAX) )
    {
      int i_m = (int)m;
      int i_n = (int)n;
      int i_lda = (int)lda;
      int i_incx = (int)incx;
      int i_incy = (int)incy;

      rocblas_handle handle = THCState_getCurrentBlasHandle(state);
      rocblas_set_stream(handle, THCState_getCurrentStream(state));
      THCublasCheck(rocblas_dger(handle, i_m, i_n, &alpha, x, i_incx, y, i_incy, a, i_lda));
      return;
    }
  THError("Cublas_Dger only supports m, n, lda, incx, incy"
          "with the bound [val] <= %d", INT_MAX);
}


rocblas_operation convertTransToCublasOperation(char trans) {
  if (trans == 't') return rocblas_operation_transpose;
  else if (trans == 'n') return rocblas_operation_none;
  else if (trans == 'c') return rocblas_operation_conjugate_transpose;
  else {
    THError("trans must be one of: t, n, c");
    return rocblas_operation_transpose;
  }
}

void adjustLdLevel3(char transa, char transb, int64_t m, int64_t n, int64_t k, int64_t *lda, int64_t *ldb, int64_t *ldc)
{
  int transa_ = ((transa == 't') || (transa == 'T'));
  int transb_ = ((transb == 't') || (transb == 'T'));

  // Note: leading dimensions generally are checked that they are > 0 and at least as big the result
  // requires (even if the value won't be used).
  if(n <= 1)
    *ldc = std::max<int64_t>(m, 1);

  if(transa_)
  {
    if(m <= 1)
      *lda = std::max<int64_t>(k, 1);
  }
  else
  {
    if(k <= 1)
      *lda = std::max<int64_t>(m, 1);
  }

  if(transb_)
  {
    if(k <= 1)
      *ldb = std::max<int64_t>(n, 1);
  }
  else
  {
    if(n <= 1)
      *ldb = std::max<int64_t>(k, 1);
  }

}

/* Level 3 */
void THCudaBlas_Sgemm(THCState *state, char transa, char transb, int64_t m, int64_t n, int64_t k, float alpha, float *a, int64_t lda, float *b, int64_t ldb, float beta, float *c, int64_t ldc)
{
  adjustLdLevel3(transa, transb, m, n, k, &lda, &ldb, &ldc);
  rocblas_operation opa = convertTransToCublasOperation(transa);
  rocblas_operation opb = convertTransToCublasOperation(transb);

  if( (m <= INT_MAX) && (n <= INT_MAX) && (k <= INT_MAX) && (lda <= INT_MAX)  && (ldb <= INT_MAX) && (ldc <= INT_MAX) )
  {
    int i_m = (int)m;
    int i_n = (int)n;
    int i_k = (int)k;
    int i_lda = (int)lda;
    int i_ldb = (int)ldb;
    int i_ldc = (int)ldc;

    rocblas_handle handle = THCState_getCurrentBlasHandle(state);
    rocblas_set_stream(handle, THCState_getCurrentStream(state));
    THCublasCheck(rocblas_sgemm(handle, opa, opb, i_m, i_n, i_k, &alpha, a, i_lda, b, i_ldb, &beta, c, i_ldc));
    return;
  }
  THError("Cublas_Sgemm only supports m, n, k, lda, ldb, ldc"
          "with the bound [val] <= %d", INT_MAX);
}

// In CUDA 8.0, definition of data types for sgemmex changed
#if CUDA_VERSION < 8000
#  define hipR16F rocblas_precision_half
#endif

void THCudaBlas_Hgemm(THCState *state, char transa, char transb, int64_t m, int64_t n, int64_t k, at::Half alpha, at::Half *a, int64_t lda, at::Half *b, int64_t ldb, at::Half beta, at::Half *c, int64_t ldc)
{
  adjustLdLevel3(transa, transb, m, n, k, &lda, &ldb, &ldc);
  rocblas_operation opa = convertTransToCublasOperation(transa);
  rocblas_operation opb = convertTransToCublasOperation(transb);

  if( (m <= INT_MAX) && (n <= INT_MAX) && (k <= INT_MAX) && (lda <= INT_MAX)  && (ldb <= INT_MAX) && (ldc <= INT_MAX) )
    {
      int i_m = (int)m;
      int i_n = (int)n;
      int i_k = (int)k;
      int i_lda = (int)lda;
      int i_ldb = (int)ldb;
      int i_ldc = (int)ldc;

      rocblas_handle handle = THCState_getCurrentBlasHandle(state);
      rocblas_set_stream(handle, THCState_getCurrentStream(state));

#ifdef __HIP_PLATFORM_HCC__
#if __hcc_workweek__ < 18451
      THCublasCheck(rocblas_hgemm(handle, opa, opb, i_m, i_n, i_k,
                    reinterpret_cast<rocblas_half*>(&alpha), reinterpret_cast<rocblas_half*>(a), i_lda,
                    reinterpret_cast<rocblas_half*>(b), i_ldb, reinterpret_cast<rocblas_half*>(&beta),
                    reinterpret_cast<rocblas_half*>(c), i_ldc));
#else
     float fAlpha = alpha;
     float fBeta = beta;
     THCublasCheck(rocblas_gemm_ex(handle, opa, opb, i_m, i_n, i_k,
                   &fAlpha, a, rocblas_datatype_f16_r, i_lda, b, rocblas_datatype_f16_r,
                   i_ldb, &fBeta, c, rocblas_datatype_f16_r, i_ldc, c, rocblas_datatype_f16_r,
                   i_ldc, rocblas_datatype_f32_r, rocblas_gemm_algo_standard, 0, 0, NULL, NULL));
#endif
#else

      // Simulated Hgemm
      float fAlpha = alpha;
      float fBeta = beta;

#if CUDA_VERSION < 9000
      THCublasCheck(rocblas_status_internal_error);
#else
      hipDeviceProp_t* prop = THCState_getCurrentDeviceProperties(state);
      if (prop->major >= 5){
        THCublasCheck(rocblas_set_math_mode(handle, HIPBLAS_TENSOR_OP_MATH));
	THCublasCheck(rocblas_gemmex(handle, opa, opb,
                                   i_m, i_n, i_k, &fAlpha,
                                   a, hipR16F, i_lda, b, hipR16F,
                                   i_ldb, &fBeta, c, hipR16F, i_ldc,
                                   hipR32F, CUBLAS_GEMM_DFALT_TENSOR_OP));
	THCublasCheck(rocblas_set_math_mode(handle, HIPBLAS_DEFAULT_MATH));
      }else{
        THCublasCheck(rocblas_status_internal_error);
      }
#endif
#endif
      return;
    }
  THError("Cublas_Hgemm only supports m, n, k, lda, ldb, ldc"
          "with th bound [val] <= %d", INT_MAX);
}

void THCudaBlas_Dgemm(THCState *state, char transa, char transb, int64_t m, int64_t n, int64_t k, double alpha, double *a, int64_t lda, double *b, int64_t ldb, double beta, double *c, int64_t ldc)
{
  adjustLdLevel3(transa, transb, m, n, k, &lda, &ldb, &ldc);
  rocblas_operation opa = convertTransToCublasOperation(transa);
  rocblas_operation opb = convertTransToCublasOperation(transb);

  if( (m <= INT_MAX) && (n <= INT_MAX) && (k <= INT_MAX) && (lda <= INT_MAX)  && (ldb <= INT_MAX) && (ldc <= INT_MAX) )
  {
    int i_m = (int)m;
    int i_n = (int)n;
    int i_k = (int)k;
    int i_lda = (int)lda;
    int i_ldb = (int)ldb;
    int i_ldc = (int)ldc;

    rocblas_handle handle = THCState_getCurrentBlasHandle(state);
    rocblas_set_stream(handle, THCState_getCurrentStream(state));
    THCublasCheck(rocblas_dgemm(handle, opa, opb, i_m, i_n, i_k, &alpha, a, i_lda, b, i_ldb, &beta, c, i_ldc));
    return;
  }
  THError("Cublas_Dgemm only supports m, n, k, lda, ldb, ldc"
          "with the bound [val] <= %d", INT_MAX);
}

#if CUDA_VERSION >= 9010  || (defined __HIP_PLATFORM_HCC__ && __hcc_workweek__ > 18451)
void THCudaBlas_HgemmStridedBatched(THCState *state, char transa, char transb, int64_t m, int64_t n, int64_t k,
                             at::Half alpha, const at::Half *a, int64_t lda, int64_t strideA, const at::Half *b, int64_t ldb, int64_t strideB,
                             at::Half beta, at::Half *c, int64_t ldc, int64_t strideC, int64_t batchCount)
{
  if( (m >= INT_MAX) || (n >= INT_MAX) || (k >= INT_MAX) || (lda >= INT_MAX)  || (ldb >= INT_MAX) || (ldc >= INT_MAX) || (batchCount >= INT_MAX) )

  {
    THError("Cublas_SgemmStridedBatched only supports m, n, k, lda, ldb, ldc, batchCount"
            "with the bound [val] <= %d", INT_MAX);
  }

  adjustLdLevel3(transa, transb, m, n, k, &lda, &ldb, &ldc);
  rocblas_operation opa = convertTransToCublasOperation(transa);
  rocblas_operation opb = convertTransToCublasOperation(transb);

  rocblas_handle handle = THCState_getCurrentBlasHandle(state);
  rocblas_set_stream(handle, THCState_getCurrentStream(state));
  float fAlpha = alpha;
  float fBeta = beta;
#ifdef __HIP_PLATFORM_HCC__
  THCublasCheck(rocblas_gemm_strided_batched_ex(handle, opa, opb, (int)m, (int)n, (int)k,
                                   (void*)&fAlpha, a, rocblas_datatype_f16_r, (int)lda, strideA,
                                   b, rocblas_datatype_f16_r, (int)ldb, strideB,
                                   (void*)&fBeta, c, rocblas_datatype_f16_r, (int)ldc, strideC,
                                   c, rocblas_datatype_f16_r, (int)ldc, strideC,
                                   (int) batchCount, rocblas_datatype_f32_r, rocblas_gemm_algo_standard,
                                   0, 0, NULL, NULL));
#else
  THCublasCheck(rocblas_set_math_mode(handle, HIPBLAS_TENSOR_OP_MATH));
  THCublasCheck(hipblasGemmStridedBatchedEx(handle,
                                   opa, opb, (int)m, (int)n, (int)k,
                                   (void*)&fAlpha, a, hipR16F, (int)lda, strideA,
                                   b, hipR16F, (int)ldb, strideB,
                                   (void*)&fBeta, c, hipR16F, (int)ldc, strideC,
                                   (int)batchCount, hipR32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP));
  THCublasCheck(rocblas_set_math_mode(handle, HIPBLAS_DEFAULT_MATH));
#endif
}
#endif

void THCudaBlas_SgemmBatched(THCState *state, char transa, char transb, int64_t m, int64_t n, int64_t k,
                             float alpha, const float *a[], int64_t lda, const float *b[], int64_t ldb,
                             float beta, float *c[], int64_t ldc, int64_t batchCount)
{
  if( (m >= INT_MAX) || (n >= INT_MAX) || (k >= INT_MAX) || (lda >= INT_MAX)  || (ldb >= INT_MAX) || (ldc >= INT_MAX) || (batchCount >= INT_MAX) )
  {
    THError("Cublas_SgemmBatched only supports m, n, k, lda, ldb, ldc, batchCount"
            "with the bound [val] <= %d", INT_MAX);
  }

#ifdef __HIP_PLATFORM_HCC__

  const int64_t stridea = (transa == 'N' || transa == 'n') ? lda*k : lda*n;
  const int64_t strideb = (transb == 'N' || transb == 'n') ? ldb*n : ldb*k;
  const int64_t stridec = ldc*n;

  THCudaBlas_SgemmStridedBatched(state, transa, transb, m, n, k, alpha, *a, lda, stridea, *b, ldb, strideb, beta, *c, ldc, stridec, batchCount);

#else

  adjustLdLevel3(transa, transb, m, n, k, &lda, &ldb, &ldc);
  rocblas_operation opa = convertTransToCublasOperation(transa);
  rocblas_operation opb = convertTransToCublasOperation(transb);

  rocblas_handle handle = THCState_getCurrentBlasHandle(state);
  rocblas_set_stream(handle, THCState_getCurrentStream(state));
  THCublasCheck(rocblas_sgemm_batched(handle,
                                   opa, opb, (int)m, (int)n, (int)k,
                                   &alpha, a, (int)lda, b, (int)ldb, &beta, c, (int)ldc,
                                   (int)batchCount));
#endif
}

#if CUDA_VERSION >= 8000 || defined __HIP_PLATFORM_HCC__
void THCudaBlas_SgemmStridedBatched(THCState *state, char transa, char transb, int64_t m, int64_t n, int64_t k,
                             float alpha, const float *a, int64_t lda, int64_t strideA, const float *b, int64_t ldb, int64_t strideB,
                             float beta, float *c, int64_t ldc, int64_t strideC, int64_t batchCount)
{
  if( (m >= INT_MAX) || (n >= INT_MAX) || (k >= INT_MAX) || (lda >= INT_MAX)  || (ldb >= INT_MAX) || (ldc >= INT_MAX) || (batchCount >= INT_MAX) )

  {
    THError("Cublas_SgemmStridedBatched only supports m, n, k, lda, ldb, ldc, batchCount"
            "with the bound [val] <= %d", INT_MAX);
  }

  adjustLdLevel3(transa, transb, m, n, k, &lda, &ldb, &ldc);
  rocblas_operation opa = convertTransToCublasOperation(transa);
  rocblas_operation opb = convertTransToCublasOperation(transb);

  rocblas_handle handle = THCState_getCurrentBlasHandle(state);
  rocblas_set_stream(handle, THCState_getCurrentStream(state));
  THCublasCheck(rocblas_sgemm_strided_batched(handle,
                                   opa, opb, (int)m, (int)n, (int)k,
                                   &alpha, a, (int)lda, strideA, b, (int)ldb, strideB, &beta, c, (int)ldc, strideC,
                                   (int)batchCount));
}
#endif

void THCudaBlas_DgemmBatched(THCState *state, char transa, char transb, int64_t m, int64_t n, int64_t k,
                             double alpha, const double *a[], int64_t lda, const double *b[], int64_t ldb,
                             double beta, double *c[], int64_t ldc, int64_t batchCount)
{
  if( (m >= INT_MAX) || (n >= INT_MAX) || (k >= INT_MAX) || (lda >= INT_MAX)  || (ldb >= INT_MAX) || (ldc >= INT_MAX) || (batchCount >= INT_MAX) )
  {
    THError("Cublas_DgemmBatched only supports m, n, k, lda, ldb, ldc, batchCount"
            "with the bound [val] <= %d", INT_MAX);
  }

#ifdef __HIP_PLATFORM_HCC__

  const int64_t stridea = (transa == 'N' || transa == 'n') ? lda*k : lda*n;
  const int64_t strideb = (transb == 'N' || transb == 'n') ? ldb*n : ldb*k;
  const int64_t stridec = ldc*n;

  THCudaBlas_DgemmStridedBatched(state, transa, transb, m, n, k, alpha, *a, lda, stridea, *b, ldb, strideb, beta, *c, ldc, stridec, batchCount);

#else

  adjustLdLevel3(transa, transb, m, n, k, &lda, &ldb, &ldc);
  rocblas_operation opa = convertTransToCublasOperation(transa);
  rocblas_operation opb = convertTransToCublasOperation(transb);

  rocblas_handle handle = THCState_getCurrentBlasHandle(state);
  rocblas_set_stream(handle, THCState_getCurrentStream(state));
  THCublasCheck(rocblas_dgemm_batched(handle,
                                   opa, opb, (int)m, (int)n, (int)k,
                                   &alpha, a, (int)lda, b, (int)ldb, &beta, c, (int)ldc,
                                   (int)batchCount));
#endif
}

#if CUDA_VERSION >= 8000 || defined __HIP_PLATFORM_HCC__
void THCudaBlas_DgemmStridedBatched(THCState *state, char transa, char transb, int64_t m, int64_t n, int64_t k,
                             double alpha, const double *a, int64_t lda, int64_t strideA, const double *b, int64_t ldb, int64_t strideB,
                             double beta, double *c, int64_t ldc, int64_t strideC, int64_t batchCount)
{
  if( (m >= INT_MAX) || (n >= INT_MAX) || (k >= INT_MAX) || (lda >= INT_MAX)  || (ldb >= INT_MAX) || (ldc >= INT_MAX) || (batchCount >= INT_MAX) )
  {
    THError("Cublas_DgemmBatched only supports m, n, k, lda, ldb, ldc, batchCount"
            "with the bound [val] <= %d", INT_MAX);
  }

  adjustLdLevel3(transa, transb, m, n, k, &lda, &ldb, &ldc);
  rocblas_operation opa = convertTransToCublasOperation(transa);
  rocblas_operation opb = convertTransToCublasOperation(transb);

  rocblas_handle handle = THCState_getCurrentBlasHandle(state);
  rocblas_set_stream(handle, THCState_getCurrentStream(state));
  THCublasCheck(rocblas_dgemm_strided_batched(handle,
                                   opa, opb, (int)m, (int)n, (int)k,
                                   &alpha, a, (int)lda, strideA, b, (int)ldb, strideB, &beta, c, (int)ldc, strideC,
                                   (int)batchCount));
}
#endif

/* Inverse */
void THCudaBlas_Sgetrf(THCState *state, int n, float **a, int lda, int *pivot, int *info, int batchSize) {
  if( (n >= INT_MAX) || (lda >= INT_MAX) || (batchSize >= INT_MAX) )
  {
    THError("Cublas_Sgetrf only supports n, lda, batchSize"
            "with the bound [val] <= %d", INT_MAX);
  }
  rocblas_handle handle = THCState_getCurrentBlasHandle(state);
  rocblas_set_stream(handle, THCState_getCurrentStream(state));
  THCublasCheck(rocblas_status_internal_error);
}

void THCudaBlas_Dgetrf(THCState *state, int n, double **a, int lda, int *pivot, int *info, int batchSize) {
  if( (n >= INT_MAX) || (lda >= INT_MAX) || (batchSize >= INT_MAX) )
  {
    THError("Cublas_Dgetrf only supports n, lda, batchSize"
            "with the bound [val] <= %d", INT_MAX);
  }
  rocblas_handle handle = THCState_getCurrentBlasHandle(state);
  rocblas_set_stream(handle, THCState_getCurrentStream(state));
  THCublasCheck(rocblas_status_internal_error);
}

void THCudaBlas_Sgetrs(THCState *state, char transa, int n, int nrhs, const float **a, int lda, int *pivot, float **b, int ldb, int *info, int batchSize)
{
  if( (n >= INT_MAX) || (nrhs >= INT_MAX) || (lda >= INT_MAX) || (ldb >= INT_MAX) || (batchSize >= INT_MAX) )
  {
    THError("Cublas_Dgetrs only supports n, nrhs, lda, ldb, batchSize"
            "with the bound [val] <= %d", INT_MAX);
  }

  // no need to adjust leading dimensions, since matrices are square
  rocblas_operation opa = convertTransToCublasOperation(transa);

  rocblas_handle handle = THCState_getCurrentBlasHandle(state);
  rocblas_set_stream(handle, THCState_getCurrentStream(state));
  THCublasCheck(rocblas_status_internal_error);
}


void THCudaBlas_Dgetrs(THCState *state, char transa, int n, int nrhs, const double **a, int lda, int *pivot, double **b, int ldb, int *info, int batchSize)
{
  if( (n >= INT_MAX) || (nrhs >= INT_MAX) || (lda >= INT_MAX) || (ldb >= INT_MAX) || (batchSize >= INT_MAX) )
  {
    THError("Cublas_Dgetrs only supports n, nrhs, lda, ldb, batchSize"
            "with the bound [val] <= %d", INT_MAX);
  }

  // no need to adjust leading dimensions, since matrices are square
  rocblas_operation opa = convertTransToCublasOperation(transa);

  rocblas_handle handle = THCState_getCurrentBlasHandle(state);
  rocblas_set_stream(handle, THCState_getCurrentStream(state));
  THCublasCheck(rocblas_status_internal_error);
}

void THCudaBlas_Sgetri(THCState *state, int n, const float **a, int lda, int *pivot, float **c, int ldc, int *info, int batchSize) {

  if( (n >= INT_MAX) || (lda >= INT_MAX)|| (ldc >= INT_MAX) || (batchSize >= INT_MAX) )
  {
    THError("Cublas_Sgetri only supports n, lda, ldc, batchSize"
            "with the bound [val] <= %d", INT_MAX);
  }
  rocblas_handle handle = THCState_getCurrentBlasHandle(state);
  rocblas_set_stream(handle, THCState_getCurrentStream(state));
  THCublasCheck(rocblas_status_internal_error);
}

void THCudaBlas_Dgetri(THCState *state, int n, const double **a, int lda, int *pivot, double **c, int ldc, int *info, int batchSize) {

  if( (n >= INT_MAX) || (lda >= INT_MAX)|| (ldc >= INT_MAX) || (batchSize >= INT_MAX) )
  {
    THError("Cublas_Dgetri only supports n, lda, ldc, batchSize"
            "with the bound [val] <= %d", INT_MAX);
  }
  rocblas_handle handle = THCState_getCurrentBlasHandle(state);
  rocblas_set_stream(handle, THCState_getCurrentStream(state));
  THCublasCheck(rocblas_status_internal_error);
}
