#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "THCSleep.h"


__global__ void spin_kernel(int64_t cycles)
{
  // see concurrentKernels CUDA sampl
  int64_t start_clock = clock64();
  int64_t clock_offset = 0;
  while (clock_offset < cycles)
  {
    clock_offset = clock64() - start_clock;
  }
}

void THC_sleep(THCState* state, int64_t cycles)
{
  dim3 grid(1);
  dim3 block(1);
 hipLaunchKernelGGL( spin_kernel, dim3(grid), dim3(block), 0, THCState_getCurrentStream(state), static_cast<int64_t>(cycles));
  THCudaCheck(hipGetLastError());
}
